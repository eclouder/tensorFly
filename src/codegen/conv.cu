
#include <hip/hip_runtime.h>

#ifdef _WIN32
using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long
#endif
__global__ void __launch_bounds__(64) default_function_kernel0(float* __restrict__ A, float* __restrict__ W, float* __restrict__ B) {
    float B_local[64];
    __shared__ float Apad_shared[512];
    __shared__ float W_shared[512];
    float Apad_shared_local[8];
    float W_shared_local[8];
    for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
        for (int nn_c_init = 0; nn_c_init < 4; ++nn_c_init) {
            B_local[((ff_c_init * 4) + nn_c_init)] = 0.000000e+00f;
            B_local[(((ff_c_init * 4) + nn_c_init) + 32)] = 0.000000e+00f;
            B_local[(((ff_c_init * 4) + nn_c_init) + 16)] = 0.000000e+00f;
            B_local[(((ff_c_init * 4) + nn_c_init) + 48)] = 0.000000e+00f;
        }
    }
    for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
        for (int ry = 0; ry < 3; ++ry) {
            for (int rx = 0; rx < 3; ++rx) {
                __syncthreads();
                for (int ax3_inner_outer = 0; ax3_inner_outer < 2; ++ax3_inner_outer) {
                    *(float4*)(Apad_shared + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) +
                                              (ax3_inner_outer * 4))) =
                            (((((1 <= ((((int)blockIdx.z) / 14) + ry)) && (((((int)blockIdx.z) / 14) + ry) < 15))
                               && (1 <= (rx + (((int)blockIdx.z) % 14)))) && ((rx + (((int)blockIdx.z) % 14)) < 15))
                             ? *(float4*)(A + (((((((((ry * 917504) + (((int)blockIdx.z) * 65536)) + (rx * 65536)) + (rc_outer * 2048)) + (((int)threadIdx.y) * 256)) + (((int)blockIdx.x) * 64))
                                                 + (((int)threadIdx.x) * 8)) + (ax3_inner_outer * 4)) - 983040)) : make_float4(0.000000e+00f, 0.000000e+00f, 0.000000e+00f, 0.000000e+00f));
                }
                for (int ax3_inner_outer1 = 0; ax3_inner_outer1 < 2; ++ax3_inner_outer1) {
                    *(float4*)(W_shared + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer1 * 4))) = *(float4*)(W + (((((((ry * 393216) + (rx * 131072)) + (rc_outer * 4096)) + (((int)threadIdx.y) * 512)) + (((int)blockIdx.y) * 64)) + (((int)threadIdx.x) * 8)) + (ax3_inner_outer1 * 4)));
                }
                __syncthreads();
                for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
                    for (int ax3 = 0; ax3 < 4; ++ax3) {
                        Apad_shared_local[ax3] = Apad_shared[(((rc_inner * 64) + (((int)threadIdx.x) * 4)) + ax3)];
                        Apad_shared_local[(ax3 + 4)] = Apad_shared[((((rc_inner * 64) + (((int)threadIdx.x) * 4)) + ax3) + 32)];
                    }
                    for (int ax31 = 0; ax31 < 4; ++ax31) {
                        W_shared_local[ax31] = W_shared[(((rc_inner * 64) + (((int)threadIdx.y) * 4)) + ax31)];
                        W_shared_local[(ax31 + 4)] = W_shared[((((rc_inner * 64) + (((int)threadIdx.y) * 4)) + ax31) + 32)];
                    }
                    for (int ff_c = 0; ff_c < 4; ++ff_c) {
                        for (int nn_c = 0; nn_c < 4; ++nn_c) {
                            B_local[((ff_c * 4) + nn_c)] = (B_local[((ff_c * 4) + nn_c)] + (Apad_shared_local[nn_c] * W_shared_local[ff_c]));
                            B_local[(((ff_c * 4) + nn_c) + 32)] = (B_local[(((ff_c * 4) + nn_c) + 32)] + (Apad_shared_local[nn_c] * W_shared_local[(ff_c + 4)]));
                            B_local[(((ff_c * 4) + nn_c) + 16)] = (B_local[(((ff_c * 4) + nn_c) + 16)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[ff_c]));
                            B_local[(((ff_c * 4) + nn_c) + 48)] = (B_local[(((ff_c * 4) + nn_c) + 48)] + (Apad_shared_local[(nn_c + 4)] * W_shared_local[(ff_c + 4)]));
                        }
                    }
                }
            }
        }
    }
    for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
        for (int nn_inner_inner_inner = 0; nn_inner_inner_inner < 4; ++nn_inner_inner_inner) {
            B[(((((((((int)blockIdx.z) * 131072) + (((int)blockIdx.y) * 16384)) + (((int)threadIdx.y) * 1024)) + (ff_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner)] = B_local[((ff_inner_inner_inner * 4) + nn_inner_inner_inner)];
            B[((((((((((int)blockIdx.z) * 131072) + (((int)blockIdx.y) * 16384)) + (((int)threadIdx.y) * 1024)) + (ff_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 8192)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 32)];
            B[((((((((((int)blockIdx.z) * 131072) + (((int)blockIdx.y) * 16384)) + (((int)threadIdx.y) * 1024)) + (ff_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 32)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 16)];
            B[((((((((((int)blockIdx.z) * 131072) + (((int)blockIdx.y) * 16384)) + (((int)threadIdx.y) * 1024)) + (ff_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 4)) + nn_inner_inner_inner) + 8224)] = B_local[(((ff_inner_inner_inner * 4) + nn_inner_inner_inner) + 48)];
        }
    }
}


